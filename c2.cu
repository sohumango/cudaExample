#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>

#define BLOCK 16
#define WIDTH 1024

float h_A[WIDTH * WIDTH];
float h_B[WIDTH * WIDTH];
float h_C[WIDTH * WIDTH]; 

float *d_A, *d_B, *d_C;

__global__ void d_multiply0(float *A, float *B, float *C)
{
    unsigned int r = blockDim.y * blockIdx.y + threadIdx.y;
    unsigned int c = blockDim.x * blockIdx.x + threadIdx.x;
    unsigned int i;
    float tmp;
    tmp = 0.0f;
    if ((r < WIDTH) && (c < WIDTH)) { 
        for (i = 0; i < WIDTH; i++)
            tmp += A[WIDTH * r + i] * B[WIDTH * i + c];
        C[WIDTH * r + c] = tmp;
    }
}

int main()
{
    unsigned int i;
    clock_t s1, e1;
    s1 = clock();
    for (i = 0; i < (WIDTH * WIDTH); i++) {
        h_A[i] = (float)i;
        h_B[i] = (float)i;
    }
    e1 = clock();
    printf("set value cost =%f\n", ((double) (e1 - s1)) / CLOCKS_PER_SEC);

    s1 = clock();
    hipMalloc((void**)&d_A, sizeof(float) * WIDTH * WIDTH);
    e1 = clock();
    printf("cudaMalloc_1 cost =%f\n", ((double) (e1 - s1)) / CLOCKS_PER_SEC);
    s1 = clock();
    hipMalloc((void**)&d_B, sizeof(float) * WIDTH * WIDTH);
    hipMalloc((void**)&d_C, sizeof(float) * WIDTH * WIDTH);
    e1 = clock();
    printf("cudaMalloc_2_3 cost =%f\n", ((double) (e1 - s1)) / CLOCKS_PER_SEC);

    s1 = clock();
    hipMemcpy(d_A, h_A, sizeof(float) * WIDTH * WIDTH, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, sizeof(float) * WIDTH * WIDTH, hipMemcpyHostToDevice);
    e1 = clock();
    printf("hipMemcpy cost =%f\n", ((double) (e1 - s1)) / CLOCKS_PER_SEC);

    dim3 grid(WIDTH / BLOCK + 1, WIDTH / BLOCK + 1);
    dim3 block(BLOCK, BLOCK);

    clock_t start, end;
    double cpu_time_used;
    start = clock();
    d_multiply0 <<< grid, block >>> (d_A, d_B, d_C);
    end = clock();
    cpu_time_used = ((double) (end - start)) / CLOCKS_PER_SEC;

    s1 = clock();
    hipMemcpy(h_C, d_C, sizeof(float) * WIDTH * WIDTH, hipMemcpyDeviceToHost);
    e1 = clock();
    printf("cudaMemcpy2 cost =%f\n", ((double) (e1 - s1)) / CLOCKS_PER_SEC);

    s1 = clock();
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    e1 = clock();
    printf("hipFree cost =%f\n", ((double) (e1 - s1)) / CLOCKS_PER_SEC);

    printf("　device result: %f, time cost = %f second\n", h_C[WIDTH * WIDTH - 1], cpu_time_used);
    return 0;
}
