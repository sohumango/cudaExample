
#include <hip/hip_runtime.h>
#include <stdio.h>
#include<stdlib.h>

dim3 grid(10, 10);
dim3 block(10,10,1);

#define BLOCK 16
#define N 200

__global__
void sum_of_array(float *arr1, float *arr2, float *arr3){
    printf("blockIdx.x = %d, blockIdx.y = %d, blockIdx.z = %d\n", blockIdx.x, blockIdx.y, blockIdx.z);
    printf("threadIdx.x = %d\n", threadIdx.x);

    int i = blockIdx.x * blockDim.x + threadIdx.x;
    arr3[i] = arr1[i] + arr2[i];
}

void initialize_array(float *arr, int size){
    for (int i = 0; i < size; i++){
        arr[i] = i*1.0;//(float)rand();
    }
}

int main(void){
    float *arr1, *arr2, *arr3, *d_arr1, *d_arr2, *d_arr3;
    size_t n_byte = N * sizeof(float);

    arr1 = (float *)malloc(n_byte);
    arr2 = (float *)malloc(n_byte);
    arr3 = (float *)malloc(n_byte);

    initialize_array(arr1, N);
    initialize_array(arr2, N);
    initialize_array(arr3, N);

    printf("start cudaMalloc\n");
    hipMalloc((void**)&d_arr1, n_byte);
    hipMalloc((void**)&d_arr2, n_byte);
    hipMalloc((void**)&d_arr3, n_byte);
    printf("finish cudaMalloc\n");

    printf("start cudaMemcpy\n");
    hipMemcpy(d_arr1, arr1, n_byte, hipMemcpyHostToDevice);
    hipMemcpy(d_arr2, arr2, n_byte, hipMemcpyHostToDevice);
    hipMemcpy(d_arr3, arr3, n_byte, hipMemcpyHostToDevice);
    printf("finish cudaMemcpy\n");

    printf("start kernel function\n");
    sum_of_array<<<(N+255)/256, 256>>>(d_arr1, d_arr2, d_arr3);
    printf("finish kernel function\n");
    hipMemcpy(arr3, d_arr3, n_byte, hipMemcpyDeviceToHost);
    hipFree(d_arr3);
    hipFree(d_arr2);
    hipFree(d_arr1);

    for(int i = 0; i <N; i++){
        printf("%f+%f = %f, ", arr1[i], arr2[i], arr3[i]);
    }
    printf("\n");
}